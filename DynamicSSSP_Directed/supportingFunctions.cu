#ifndef SUPPORTING_CU
#define SUPPORTING_CU

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <vector>
#include <chrono>

#include "all_structure_dir.cuh"
#include "gpuFunctions_dir.cuh"
//#include "bfs.cu"
using namespace std;
using namespace std::chrono;


void transfer_data_to_GPU(vector<ColWtList>& InEdgesList, int*& InEdgesListTracker, vector<ColWt>& InEdgesListFull, ColWt*& InEdgesListFull_device, int*& InEdgesListTracker_device, vector<ColWtList>& AdjList, int*& AdjListTracker, vector<ColWt>& AdjListFull, ColWt*& AdjListFull_device,
	int nodes, int edges, int totalInsertion, int*& AdjListTracker_device, bool zeroInsFlag,
	vector<changeEdge>& allChange_Ins, changeEdge*& allChange_Ins_device, int totalChangeEdges_Ins,
	int deviceId, int totalChangeEdges_Del, bool zeroDelFlag, changeEdge*& allChange_Del_device,
	int*& counter_del, int*& affectedNodeList_del, int*& updatedAffectedNodeList_del, int*& updated_counter_del, vector<changeEdge>& allChange_Del, size_t  numberOfBlocks)
{
	hipError_t cudaStatus;

	//create 1D array from 2D to fit it in GPU
	cout << "creating 1D array from 2D to fit it in GPU" << endl;
	AdjListTracker[0] = 0; //start pointer points to the first index of InEdgesList
	for (int i = 0; i < nodes; i++) {
		AdjListTracker[i + 1] = AdjListTracker[i] + AdjList.at(i).size();
		AdjListFull.insert(std::end(AdjListFull), std::begin(AdjList.at(i)), std::end(AdjList.at(i)));
	}
	InEdgesListTracker[0] = 0; //start pointer points to the first index of InEdgesList
	for (int i = 0; i < nodes; i++) {
		InEdgesListTracker[i + 1] = InEdgesListTracker[i] + InEdgesList.at(i).size();
		InEdgesListFull.insert(std::end(InEdgesListFull), std::begin(InEdgesList.at(i)), std::end(InEdgesList.at(i)));
	}
	cout << "creating 1D array from 2D completed" << endl;


	//Transferring input graph and change edges data to GPU
	cout << "Transferring graph data from CPU to GPU" << endl;
	auto startTime_transfer = high_resolution_clock::now();
	printf("edges: %d totalInsertion:%d sizeof(ColWt):%d \n", edges, totalInsertion, sizeof(ColWt));
	cudaStatus = hipMallocManaged(&AdjListFull_device, (edges + totalInsertion) * sizeof(ColWt));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at InEdgesListFull structure");
	}
	//printf("testA1");
	std::copy(AdjListFull.begin(), AdjListFull.end(), AdjListFull_device);
	//printf("testA2");

	cudaStatus = hipMalloc((void**)&AdjListTracker_device, (nodes + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at InEdgesListTracker_device");
	}
	hipMemcpy(AdjListTracker_device, AdjListTracker, (nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
	//printf("testB");
	//Asynchronous prefetching of data
	hipMemPrefetchAsync(AdjListFull_device, edges * sizeof(ColWt), deviceId);
	//printf("testC");





	cudaStatus = hipMallocManaged(&InEdgesListFull_device, (edges + totalInsertion) * sizeof(ColWt));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at InEdgesListFull structure");
	}
	//printf("testA1");
	std::copy(InEdgesListFull.begin(), InEdgesListFull.end(), InEdgesListFull_device);
	//printf("testA2");

	cudaStatus = hipMalloc((void**)&InEdgesListTracker_device, (nodes + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at InEdgesListTracker_device");
	}
	hipMemcpy(InEdgesListTracker_device, InEdgesListTracker, (nodes + 1) * sizeof(int), hipMemcpyHostToDevice);
	//printf("testB");
	//Asynchronous prefetching of data
	hipMemPrefetchAsync(InEdgesListFull_device, edges * sizeof(ColWt), deviceId);






	if (zeroInsFlag != true) {
		cudaStatus = hipMallocManaged(&allChange_Ins_device, totalChangeEdges_Ins * sizeof(changeEdge));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed at allChange_Ins structure");
		}
		std::copy(allChange_Ins.begin(), allChange_Ins.end(), allChange_Ins_device);
		//printf("testD");
		//set hipMemAdviseSetReadMostly by the GPU for change edge data
		hipMemAdvise(allChange_Ins_device, totalChangeEdges_Ins * sizeof(changeEdge), hipMemAdviseSetReadMostly, deviceId);
		//printf("testE");
		//Asynchronous prefetching of data
		hipMemPrefetchAsync(allChange_Ins_device, totalChangeEdges_Ins * sizeof(changeEdge), deviceId);
		//printf("testF");
	}

	if (zeroDelFlag != true) {
		cudaStatus = hipMallocManaged(&allChange_Del_device, totalChangeEdges_Del * sizeof(changeEdge));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed at allChange_Del structure");
		}
		std::copy(allChange_Del.begin(), allChange_Del.end(), allChange_Del_device);
		//set hipMemAdviseSetReadMostly by the GPU for change edge data
		hipMemAdvise(allChange_Del_device, totalChangeEdges_Del * sizeof(changeEdge), hipMemAdviseSetReadMostly, deviceId);
		//Asynchronous prefetching of data
		hipMemPrefetchAsync(allChange_Del_device, totalChangeEdges_Del * sizeof(changeEdge), deviceId);

		counter_del = 0;
		hipMallocManaged(&counter_del, sizeof(int));
		hipMallocManaged(&affectedNodeList_del, nodes * sizeof(int));
		hipMallocManaged(&updatedAffectedNodeList_del, nodes * sizeof(int));
		updated_counter_del = 0;
		hipMallocManaged(&updated_counter_del, sizeof(int));

		//modify adjacency list to adapt the deleted edges
		deleteEdgeFromAdj << < numberOfBlocks, THREADS_PER_BLOCK >> > (allChange_Del_device, totalChangeEdges_Del, InEdgesListFull_device, InEdgesListTracker_device, AdjListFull_device, AdjListTracker_device);
		hipDeviceSynchronize();
	}



	auto stopTime_transfer = high_resolution_clock::now();//Time calculation ends
	auto duration_transfer = duration_cast<microseconds>(stopTime_transfer - startTime_transfer);// duration calculation
	cout << "**Time taken to transfer graph data from CPU to GPU: "
		<< float(duration_transfer.count()) / 1000 << " milliseconds**" << endl;
}

void read_and_transfer_input_SSSPtree_to_GPU(char* inputSSSPfile, vector<ColList>& SSSPTreeAdjList, int*& SSSPTreeAdjListTracker, vector<int>& SSSPTreeAdjListFull,
	RT_Vertex*& SSSP, int nodes, int edges, int*& SSSPTreeAdjListFull_device, int*& SSSPTreeAdjListTracker_device, /*vector<int>& hop,*/ int deviceId/*, int*& d_hop*/)
{
	hipError_t cudaStatus;

	SSSPTreeAdjList.resize(nodes);
	SSSPTreeAdjListTracker = (int*)malloc((nodes + 1) * sizeof(int));//we take nodes +1 to store the start ptr of the first row

	cudaStatus = hipMallocManaged(&SSSP, nodes * sizeof(RT_Vertex));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSP structure");
	}
	cout << "Reading input SSSP tree data..." << endl;
	auto readSSSPstartTime = high_resolution_clock::now();//Time calculation starts
	read_SSSP(SSSP, inputSSSPfile, &nodes, SSSPTreeAdjList);


	//New addition
	SSSPTreeAdjListTracker[0] = 0; //start pointer points to the first index of InEdgesList
	for (int i = 0; i < nodes; i++) {
		SSSPTreeAdjListTracker[i + 1] = SSSPTreeAdjListTracker[i] + SSSPTreeAdjList.at(i).size();
		SSSPTreeAdjListFull.insert(std::end(SSSPTreeAdjListFull), std::begin(SSSPTreeAdjList.at(i)), std::end(SSSPTreeAdjList.at(i)));
	}


	//Transferring SSSP tree data to GPU

	cudaStatus = hipMallocManaged(&SSSPTreeAdjListFull_device, (nodes) * sizeof(int)); //new change to nodes from nodes -1 as 0 0 0 is also a row in SSSP file//SSSP tree has n-1 edges and we consider each edge 1 time
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSPTreeAdjListFull_device structure");
	}
	std::copy(SSSPTreeAdjListFull.begin(), SSSPTreeAdjListFull.end(), SSSPTreeAdjListFull_device);


	cudaStatus = hipMalloc((void**)&SSSPTreeAdjListTracker_device, (nodes + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed at SSSPTreeAdjListTracker_device");
	}
	hipMemcpy(SSSPTreeAdjListTracker_device, SSSPTreeAdjListTracker, (nodes + 1) * sizeof(int), hipMemcpyHostToDevice);


	//compute hop
	//vector<bool> visited;
	//int startVertex = 0; //0 is considered as root vertex
	//visited = vector<bool>(nodes);
	////hop = vector<int>(nodes);

	//const int size = nodes * sizeof(int);
	//hipMalloc((void**)&d_hop, size);
	//hop = vector<int>(nodes, INT_MAX);
	//hop[startVertex] = 0;
	//hipMemcpy(d_hop, hop.data(), size, hipMemcpyHostToDevice);

	//??we don't need this hop computing now
	/*auto startTime = chrono::steady_clock::now();
	bfsGPU(startVertex, nodes, SSSPTreeAdjListFull_device, SSSPTreeAdjListTracker_device, d_hop, visited);
	auto endTime = std::chrono::steady_clock::now();
	long duration = chrono::duration_cast<chrono::milliseconds>(endTime - startTime).count();
	printf("Elapsed time for hop computation : %li ms.\n", duration);*/

	//hipDeviceSynchronize();
	//hipMemcpy(&hop[0], d_hop, size, hipMemcpyDeviceToHost);

	auto readSSSPstopTime = high_resolution_clock::now();//Time calculation ends
	auto readSSSPduration = duration_cast<microseconds>(readSSSPstopTime - readSSSPstartTime);// duration calculation
	cout << "Reading input SSSP tree data completed" << endl;
	cout << "Time taken to read input input SSSP tree: " << readSSSPduration.count() << " microseconds" << endl;
	//set hipMemAdviseSetPreferredLocation at GPU for SSSP data
	hipMemAdvise(SSSP, nodes * sizeof(RT_Vertex), hipMemAdviseSetPreferredLocation, deviceId);

	//Asynchronous prefetching of data
	hipMemPrefetchAsync(SSSP, nodes * sizeof(RT_Vertex), deviceId);

}
#endif